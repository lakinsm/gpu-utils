#include <hip/hip_runtime.h>
#include <iostream>
#include <exception>


void cuda_error(hipError_t e, int code_line) {
    if(e != hipSuccess) {
        std::cerr << "CUDA execution error: " << e << " at line " << code_line << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(e) (cuda_error(e, __LINE__))


// Function to gather information about devices prior to memory allocation
void print_cuda_device_properties() {
    hipDeviceProp_t prop;
    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));
    for(int i = 0; i < count; ++i) {
        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        std::cout << "===== Graphics Device Information: GPU" << i << " =====    " << std::endl;
        std::cout << "Name: " << prop.name << std::endl;
        std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "Clock rate: " << prop.clockRate << std::endl;
        std::cout << "Device copy overlap: ";
        if(prop.deviceOverlap)
            std::cout << "Enabled" << std::endl;
        else
            std::cout << "Disabled" << std::endl;
        std::cout << "Kernel execition timeout: ";
        if(prop.kernelExecTimeoutEnabled)
            std::cout << "Enabled" << std::endl << std::endl;
        else
            std::cout << "Disabled" << std::endl << std::endl;

        std::cout << "===== Graphics Device Memory Information: GPU #" << i << " =====    " << std::endl;
        std::cout << "Total global memory: " << (float)prop.totalGlobalMem / 1000000000 << " GB" << std::endl;
        std::cout << "Total constant memory: " << (float)prop.totalConstMem / 1000 << " KB" << std::endl;
        std::cout << "Max memory pitch: " << (float)prop.memPitch / 1000000000 << " GB" << std::endl;
        std::cout << "Texture alignment: " << prop.textureAlignment << std::endl << std::endl;

        std::cout << "===== Graphics Device MultiProcessor Information: GPU #" << i << " =====    " << std::endl;
        std::cout << "Multiprocessor count: " << prop.multiProcessorCount << std::endl;
        std::cout << "Shared memory per MP: " << (float)prop.sharedMemPerBlock / 1000 << " KB" << std::endl;
        std::cout << "Registers per MP: " << prop.regsPerBlock << std::endl;
        std::cout << "Threads in warp: " << prop.warpSize << std::endl;
        std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "Max thread dimensions: " << prop.maxThreadsDim[0] << ' ' << prop.maxThreadsDim[1] << ' ';
        std::cout << prop.maxThreadsDim[2] << std::endl;
        std::cout << "Max grid dimensions: " << prop.maxGridSize[0] << ' ' << prop.maxGridSize[1] << ' ';
        std::cout << prop.maxGridSize[2] << std::endl << std::endl << std::endl << std::endl;
    }

}

int main() {
    print_cuda_device_properties();
    return 0;
}

